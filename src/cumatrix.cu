#include "hip/hip_runtime.h"
#include "cumatrix.h"
#include <limits>

double ONE = 1.0f;
double NEG_ONE = -1.0f;
double HALF = 0.5f;
double NEG_HALF = -0.5f;
double ZERO = 0.0f;
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
printf("Error at %s:%d\n",__FILE__,__LINE__);\
exit(EXIT_FAILURE);}} while(0) 

__global__ void subKernel(double *origin, double*sub, int rs, int re, int cs,
		int ce, int m);
__global__ void eTimesSPKernel(int*row, int*col, double*val, double*d, int m,
		int n);
__global__ void etimesKernel(int*row1, int*col1, double*v1, int*row2, int*col2,
		double*v2, double*d, int m, int n);
__global__ void rowNormSPKernel(int*row, double*val, double*r, int m);
__global__ void rowMulKernel(int*row, double*val, double alpha, int m);
__global__ void rowSumKernel(int*row, double*val, int m, double*res);
//__global__ void eTimesKernel(double*va, const double*vb, int n);
//__global__ void eDivKernel(double*va, const double*vb, int n);
__global__ void signPlusKernel(double*d, double*s, double v, int n);
__global__ void timesDiagKernel(double*r, double*A, double*d, int m, int n,
		int k, double alpha, bool trans);
//__global__ void sqrtKernel(double* v, int n);
__global__ void projectKernel(double* v, int n);
__global__ void shrinkKernel(double tau, double* d, int m, int n);
__global__ void repmatKernel(double*r, double*d, int mm, int mn, int m, int n);
__global__ void subvecKernel(int*row, int*col, int r, double*v, double*d);
__global__ void getRowKernel(double*v, double*d, int r, int m, int n);
__global__ void setRowKernel(double*v, double*d, int r, int m, int n);
__global__ void plusDiagKernel(double*A, double*d, double a, double b, int m,
		int n);
__global__ void rbindKernel(double*r, double*d1, double*d2, int m1, int m2,
		int n, double a, double b);

void Dense::clean() {
	if (cu_val)
		checkCudaErrors(hipFree(cu_val));
	cu_val = 0;
}

void Dense::clone(Dense*d) {
	d->clean();
	d->initial(m, n);
	int len = length();
	checkCudaErrors(hipblasDcopy(handle, len, cu_val, 1, d->cu_val, 1));
	//hipMemcpy(d->cu_val, cu_val, sizeof(float)*m*n, hipMemcpyDeviceToDevice);
}

void Dense::copyto(Dense* d) {
	int len = length();
	checkCudaErrors(hipblasDcopy(handle, len, cu_val, 1, d->cu_val, 1));
}

double Dense::dot(Dense* d) {
	int n = length();
	double res;
	checkCudaErrors(hipblasDdot(handle, n, cu_val, 1, d->cu_val, 1, &res));
	return res;
}

void Dense::ger(Dense*x, Dense*y, double a) {
	checkCudaErrors(
			hipblasDger(handle, m, n, &a, x->cu_val, 1, y->cu_val, 1, cu_val, m));
}

void Dense::initial(int m, int n) {
	this->m = m;
	this->n = n;
	//printf("m=%d,n=%d\n", m, n);
	checkCudaErrors(hipMalloc((void** ) &cu_val, sizeof(double) * m * n));
	val = thrust::device_pointer_cast(cu_val);
}

void Dense::input(const char* filename) {
	FILE* file = fopen(filename, "r");
	fscanf(file, "%d %d", &m, &n);
	size_t size = sizeof(double) * m * n;
	double*elem = (double*) malloc(size);
	//int len = m*n;
	int i, j;
	for (i = 0; i < m; ++i)
		for (j = 0; j < n; ++j)
			fscanf(file, "%lf", &elem[i + j * m]);
	//PRINT_MATRIX(val,m,n);
	checkCudaErrors(hipMalloc((void** ) &cu_val, sizeof(double) * m * n));
	checkCudaErrors(hipMemcpy(cu_val, elem, size, hipMemcpyHostToDevice));
	val = thrust::device_pointer_cast(cu_val);
	fclose(file);
	free(elem);
}

void Dense::setRandom() {
	hiprandGenerator_t gen;
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 123ULL));
	CURAND_CALL(hiprandGenerateUniformDouble(gen, cu_val, length()));
	CURAND_CALL(hiprandDestroyGenerator(gen));
}

void Dense::setValue(double v) {
	thrust::fill(val, val + m * n, v);
}

void Dense::setDiagValue(double v) {
	int size = fminf(m, n);
	for (int i = 0; i < size; ++i)
		val[i + i * m] = v;
}

void Dense::setIdentity(int s) {
	initial(s, s);
	setValue(0.0f);
	for (int i = 0; i < s; ++i)
		val[i * s + i] = 1;
}

void Dense::setIdentity() {
	setValue(0.0f);
	setDiagValue(1.0f);
}

void Dense::setElem(int i, int j, double v) {
	val[i + j * m] = v;
}

double Dense::getElem(int i, int j) {
	return val[i + j * m];
}

void Dense::transpose(Dense* trans) {
	int m = this->n;
	int n = this->m;
	trans->clean();
	trans->initial(m, n);
	//checkCudaErrors(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));
	checkCudaErrors(
			hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &ONE, cu_val,
					this->m, &ZERO, cu_val, this->m, trans->cu_val, trans->m));
}

void Dense::transpose() {
	Dense* trans = new Dense;
	transpose(trans);
	clean();
	trans->clone(this);
	delete trans;
}

void Dense::colSum(Dense* vec) {
	Dense* tran = new Dense;
	transpose(tran);
	Dense*v = new Dense;
	tran->rowSum(v);
	v->transpose(vec);
	delete tran;
	delete v;
}

void Dense::rowSum(Dense* vec) {
	Dense* one = new Dense;
	one->initial(n, 1);
	one->setValue(1.0f);
	vec->clean();
	vec->initial(m, 1);
	checkCudaErrors(
			hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &ONE, cu_val, m, one->cu_val, 1, &ZERO, vec->cu_val, 1));
	delete one;
}

void Dense::rowSquare(Dense*r) {
	Dense*m = new Dense;
	this->clone(m);
	m->eTimes(m);
	m->rowSum(r);
	delete m;
}
void Dense::colSquare(Dense*r) {
	Dense*m = new Dense;
	this->clone(m);
	m->eTimes(m);
	m->colSum(r);
	delete m;
}

void Dense::rowNorm(Dense*r) {
	rowSquare(r);
	r->square_root();
}
void Dense::colNorm(Dense*r) {
	colSquare(r);
	r->square_root();
}

void Dense::eig(Dense*Q, Dense*D) {
	D->clean();
	D->initial(m, 1);
	clone(Q);
	int lwork;
	checkCudaErrors(
			hipsolverDnDsyevd_bufferSize(solver_handle, HIPSOLVER_EIG_MODE_VECTOR,
					HIPBLAS_FILL_MODE_UPPER, m, Q->cu_val, m, D->cu_val,
					&lwork));
//	printf("lwork=%d\n", lwork);
	double*work = NULL;
	int* devInfo = NULL;
	checkCudaErrors(hipMalloc((void** ) &work, sizeof(double) * lwork));
	checkCudaErrors(hipMalloc((void** ) &devInfo, sizeof(int)));
	hipsolverDnDsyevd(solver_handle, HIPSOLVER_EIG_MODE_VECTOR,
			HIPBLAS_FILL_MODE_UPPER, m, Q->cu_val, m, D->cu_val, work, lwork,
			devInfo);
//	printf("devInfo=%d\n", devInfo);
	if (work)
		checkCudaErrors(hipFree(work));
	if (devInfo)
		checkCudaErrors(hipFree(devInfo));
}

void Dense::eig(Dense*D) {
	D->clean();
	D->initial(m, 1);
	int lwork;
	checkCudaErrors(
			hipsolverDnDsyevd_bufferSize(solver_handle, HIPSOLVER_EIG_MODE_VECTOR,
					HIPBLAS_FILL_MODE_UPPER, m, cu_val, m, D->cu_val, &lwork));
//	printf("lwork=%d\n", lwork);
	double*work = NULL;
	int* devInfo = NULL;
	checkCudaErrors(hipMalloc((void** ) &work, sizeof(double) * lwork));
	checkCudaErrors(hipMalloc((void** ) &devInfo, sizeof(int)));
	hipsolverDnDsyevd(solver_handle, HIPSOLVER_EIG_MODE_VECTOR,
			HIPBLAS_FILL_MODE_UPPER, m, cu_val, m, D->cu_val, work, lwork,
			devInfo);
//	printf("devInfo=%d\n", devInfo);
	if (work)
		checkCudaErrors(hipFree(work));
	if (devInfo)
		checkCudaErrors(hipFree(devInfo));
}

void Dense::eig() {
	double*d = NULL;
	double*work = NULL;
	int* devInfo = NULL;
	checkCudaErrors(hipMalloc((void** ) &d, sizeof(double) * m));
	int lwork;
	checkCudaErrors(
			hipsolverDnDsyevd_bufferSize(solver_handle, HIPSOLVER_EIG_MODE_VECTOR,
					HIPBLAS_FILL_MODE_UPPER, m, cu_val, m, d, &lwork));
	checkCudaErrors(hipMalloc((void** ) &work, sizeof(double) * lwork));
	checkCudaErrors(hipMalloc((void** ) &devInfo, sizeof(int)));
	hipsolverDnDsyevd(solver_handle, HIPSOLVER_EIG_MODE_VECTOR,
			HIPBLAS_FILL_MODE_UPPER, m, cu_val, m, d, work, lwork, devInfo);
	if (work)
		checkCudaErrors(hipFree(work));
	if (devInfo)
		checkCudaErrors(hipFree(devInfo));
	if (d)
		checkCudaErrors(hipFree(d));
}

void Dense::eigs(int k, Dense*Q) {
	Q->clean();
	Q->initial(m, k);
	Q->setRandom();
	Q->orth();
	for (int i = 0; i < 10; ++i) {
		Q->ltimes(this, 1.0f, false, false);
		Q->orth();
	}
}

void Dense::orth() {
	int lwork_geqrf;
	int lwork_orgqr;
	int lwork;
	int*devInfo = NULL;
	double*work = NULL;
	double*tau = NULL;
	checkCudaErrors(hipMalloc((void** )&tau, sizeof(double) * n));
	checkCudaErrors(hipMalloc((void** )&devInfo, sizeof(int)));
	checkCudaErrors(
			hipsolverDnDgeqrf_bufferSize(solver_handle, m, n, cu_val, m,
					&lwork_geqrf));
	checkCudaErrors(
			hipsolverDnDorgqr_bufferSize(solver_handle, m, n, n, cu_val, m, tau,
					&lwork_orgqr));
	lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
	checkCudaErrors(hipMalloc((void** )&work, sizeof(double) * lwork));
	checkCudaErrors(
			hipsolverDnDgeqrf(solver_handle, m, n, cu_val, m, tau, work, lwork,
					devInfo));
	checkCudaErrors(
			hipsolverDnDorgqr(solver_handle, m, n, n, cu_val, m, tau, work,
					lwork, devInfo));
	if (devInfo)
		checkCudaErrors(hipFree(devInfo));
	if (work)
		checkCudaErrors(hipFree(work));
	if (tau)
		checkCudaErrors(hipFree(tau));
}

void Dense::inv(Dense*B) {
	if (m != n) {
		printf("not square\n");
		return;
	}
	B->clean();
	B->initial(m, m);
	B->setIdentity();
	solve(B);
}

struct reciprocal: public thrust::unary_function<double, double> {
	const double a;
	reciprocal(double _a) :
			a(_a) {
	}
	__host__ __device__
	double operator()(const double &x) const {
		if (x > a)
			return 1 / x;
		else
			return 0;
	}
};

void Dense::pinv(Dense*B, double tol) {
	Dense*U = new Dense;
	Dense*V = new Dense;
	Dense*S = new Dense;

	svd(U, V, S);
	thrust::transform(S->val, S->val + S->length(), S->val, reciprocal(tol));
	V->timesDiag(B, S, 1.0, false, U->m);
	B->rtimes(U, 1.0, false, true);
	delete U;
	delete V;
	delete S;
}

void Dense::truncation(int k) {
	if (k > m || k > n) {
		printf("k is larger than m or n\n");
		return;
	}
	Dense*U = new Dense;
	Dense*V = new Dense;
	Dense*S = new Dense;
	svd(U, V, S);
	thrust::fill(S->val + k, S->val + S->length(), 0);
	U->timesDiag(this, S, 1.0, false, V->m);
	rtimes(V, 1.0, false, true);
	delete U;
	delete V;
	delete S;
//	S->sor
}

void Dense::svd(Dense*U, Dense*V, Dense*S) {
	Dense*T = new Dense;
	if (m < n) {
		transpose(T);
		T->svd(V, U, S);
//		U->transpose();
		delete T;
		return;
	}
	clone(T);
	U->clean();
	U->initial(m, m);
	V->clean();
	V->initial(n, n);
	S->clean();
	S->initial(m, 1);
	S->setValue(0);
	int lwork = 0;
	checkCudaErrors(hipsolverDnDgesvd_bufferSize(solver_handle, m, n, &lwork));
	double*work = 0;
	double*rwork = 0;
	int*devInfo = 0;
	checkCudaErrors(hipMalloc((void** )&devInfo, sizeof(int)));
	checkCudaErrors(hipMalloc((void** )&work, sizeof(double) * lwork));
	checkCudaErrors(
			hipsolverDnDgesvd(solver_handle, 'A', 'A', m, n, T->cu_val, m,
					S->cu_val, U->cu_val, m, V->cu_val, n, work, lwork, rwork,
					devInfo));
	V->transpose();
	delete T;
}

void Dense::print() {
	printf("%d %d\n", m, n);
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++)
//			printf("%.2f ", v[i * n + j]);
			std::cout << val[i + j * m] << " ";
		std::cout << std::endl;
	}
}

void Dense::print(const char* filename) {
	std::ofstream out(filename);
	out.precision(std::numeric_limits<double>::max_digits10);
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++)
			out << val[i + j * m] << " ";
		out << std::endl;
	}
	out.close();
}

void Dense::gemv(Dense*x, Dense*y, double alpha, double beta, bool trans) {
	if (trans)
		checkCudaErrors(
				hipblasDgemv(handle, HIPBLAS_OP_T, m, n, &alpha, cu_val, m, x->cu_val, 1, &beta, y->cu_val, 1));
	else
		checkCudaErrors(
				hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha, cu_val, m, x->cu_val, 1, &beta, y->cu_val, 1));
}

void Dense::timesVec(Dense*r, Dense*v, bool trans) {
	r->clean();
	if (trans)
		r->initial(n, 1);
	else
		r->initial(m, 1);
	gemv(v, r, 1.0f, 0.0f, trans);
}

void Dense::timesDiag(Dense*r, Dense*d, double alpha, bool left, int n) {
	r->clean();
	if (left)
		r->initial(this->n, n);
	else
		r->initial(this->m, n);
	r->setValue(0);
	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
	dim3 numBlocks((r->m + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(r->n + threadsPerBlock.y - 1) / threadsPerBlock.y);
	timesDiagKernel<<<numBlocks, threadsPerBlock>>>(r->cu_val, cu_val,
			d->cu_val, r->m, r->n, left ? this->m : this->n, alpha, left);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::timesDiag(Dense*r, Dense*d, double alpha, bool left) {
	int n = left ? this->m : this->n;
	timesDiag(r, d, alpha, left, n);
//	r->clean();
//	r->initial(m, n);
//	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
//	dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
//			(n + threadsPerBlock.y - 1) / threadsPerBlock.y);
//	timesDiagKernel<<<numBlocks, threadsPerBlock>>>(r->cu_val, cu_val,
//			d->cu_val, m, n, alpha, left);
//	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::times(Dense*A, Dense*B, double alpha, double beta, bool tA,
		bool tB) {
	int k;
	if (tA) {
		k = A->m;
		if (tB)
			checkCudaErrors(
					hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &beta, A->cu_val, A->m, B->cu_val, B->m, &alpha, cu_val, m));
		else
			checkCudaErrors(
					hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &beta, A->cu_val, A->m, B->cu_val, B->m, &alpha, cu_val, m));
	} else {
		k = A->n;
		if (tB)
			checkCudaErrors(
					hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &beta, A->cu_val, A->m, B->cu_val, B->m, &alpha, cu_val, m));
		else
			checkCudaErrors(
					hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &beta, A->cu_val, A->m, B->cu_val, B->m, &alpha, cu_val, m));
	}
}
struct multiply_const: public thrust::unary_function<double, double> {
	const double a;
	multiply_const(double _a) :
			a(_a) {
	}
	__host__ __device__
	double operator()(const double &x) const {
		return x * a;
	}
};
void Dense::times(Dense*r, double a) {
	r->clean();
	r->initial(m, n);
	thrust::transform(val, val + length(), r->val, multiply_const(a));
}
void Dense::times(double a) {
	thrust::transform(val, val + length(), val, multiply_const(a));
}
struct multiply: public thrust::binary_function<double, double, double> {
	const double a;
	multiply(double _a) :
			a(_a) {
	}
	__host__ __device__
	double operator()(const double &x, const double &y) const {
		return x * y * a;
	}
};
void Dense::eTimes(Dense* d, double a) {
	thrust::transform(val, val + length(), d->val, val, multiply(a));
}

void Dense::eTimes(Dense*d) {
	eTimes(d, 1.0f);
}

void Dense::eTimes(Dense*r, Dense*d, double a) {
	r->clean();
	r->initial(m, n);
	thrust::transform(val, val + length(), d->val, r->val, multiply(a));
}

struct divide: public thrust::binary_function<double, double, double> {
	__host__ __device__
	double operator()(const double &x, const double &y) const {
		return x / y;
	}
};
void Dense::eDiv(Dense*d) {
	thrust::transform(val, val + length(), d->val, val, divide());
//	int thread = THREADS;
//	int len = length();
//	int block = (len + thread - 1) / thread;
//	eDivKernel<<<block, thread>>>(cu_val, d->cu_val, len);
//	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::rtimes(Dense*r, Dense*A, double alpha, bool trans, bool tA) {
	r->clean();
	int m, n;
	m = (trans) ? this->n : this->m;
	n = (tA) ? A->m : A->n;
	r->initial(m, n);
	r->times(this, A, 0.0f, alpha, trans, tA);
}

void Dense::rtimes(Dense*A, double alpha, bool trans, bool tA) {
	Dense*B = new Dense;
	this->clone(B);
	B->rtimes(this, A, alpha, trans, tA);
	delete B;
}

void Dense::ltimes(Dense*r, Dense*A, double alpha, bool trans, bool tA) {
	r->clean();
	int m, n;
	m = (tA) ? A->n : A->m;
	n = (trans) ? this->m : this->n;
	r->initial(m, n);
	r->times(A, this, 0.0f, alpha, tA, trans);
}

void Dense::ltimes(Dense*A, double alpha, bool trans, bool tA) {
	Dense*B = new Dense;
	this->clone(B);
	B->ltimes(this, A, alpha, trans, tA);
	delete B;
}
//void Dense::times(Dense*r, Dense* d, bool tA, bool tB) {
////	int m = this->m;
////	int k = this->n;
////	int n = d->n;
//	r->clean();
//	int m, n, k;
//	r->clean();
//	if (tA) {
//		m = this->n;
//		k = this->m;
//		if (tB) {
//			n = d->m;
//			r->initial(m, n);
//			checkCudaErrors(
//					hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &ONE, cu_val, this->m, d->cu_val, d->m, &ZERO, r->cu_val, m));
//		} else {
//			n = d->n;
//			r->initial(m, n);
//			checkCudaErrors(
//					hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &ONE, cu_val, this->m, d->cu_val, d->m, &ZERO, r->cu_val, m));
//		}
//	} else {
//		m = this->m;
//		k = this->n;
//		if (tB) {
//			n = d->m;
//			r->initial(m, n);
//			checkCudaErrors(
//					hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &ONE, cu_val, this->m, d->cu_val, d->m, &ZERO, r->cu_val, m));
//		} else {
//			n = d->n;
//			r->initial(m, n);
//			checkCudaErrors(
//					hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &ONE, cu_val, this->m, d->cu_val, d->m, &ZERO, r->cu_val, m));
//		}
//	}
//}

//void Dense::plus(Dense*addend, float a) {
//	Dense*r = new Dense;
//	plus(r, addend, 1.0f, a);
//	r->clone(this);
//	delete r;
//}
//
//void Dense::plus(Dense*addend, float a, float b) {
//	Dense*r = new Dense;
//	plus(r, addend, a, b);
//	r->clone(this);
//	delete r;
//}
//
//void Dense::plus(Dense* res, Dense* addend, float a, float b) {
//	plus(res, addend, a, b, false, false);
//}

void Dense::plus(double alpha, double beta) {
	Dense* ones = new Dense;
	ones->initial(m, n);
	ones->setValue(1.0f);
	plus(ones, alpha, beta, false);
	delete ones;
}

void Dense::plus(Dense* d, double a, double b, bool trans) {
	if (trans)
		checkCudaErrors(
				hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, &a, cu_val,
						m, &b, d->cu_val, d->m, cu_val, m));
	else
		checkCudaErrors(
				hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &a, cu_val,
						m, &b, d->cu_val, d->m, cu_val, m));
}

void Dense::plus(Dense* res, Dense* d, double a, double b, bool tA, bool tB) {
	res->clean();
	int m, n;
	if (tA) {
		m = this->n;
		n = this->m;
		res->initial(m, n);
		if (tB)
			checkCudaErrors(
					hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &a,
							cu_val, this->m, &b, d->cu_val, d->m, res->cu_val,
							m));
		else
			checkCudaErrors(
					hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &a,
							cu_val, this->m, &b, d->cu_val, d->m, res->cu_val,
							m));
	} else {
		m = this->m;
		n = this->n;
		res->initial(m, n);
		if (tB)
			checkCudaErrors(
					hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, &a,
							cu_val, this->m, &b, d->cu_val, d->m, res->cu_val,
							m));
		else
			checkCudaErrors(
					hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &a,
							cu_val, this->m, &b, d->cu_val, d->m, res->cu_val,
							m));
	}
}

void Dense::plusDiag(Dense*r, Dense*d, double a, double b, bool tran) {
	if (!tran)
		clone(r);
	else
		transpose(r);
	r->plusDiag(d, a, b);
}

void Dense::plusDiag(Dense*d, double a, double b) {
	int thread = THREADS;
	int block = (m + thread - 1) / thread;
	plusDiagKernel<<<block, thread>>>(cu_val, d->cu_val, a, b, m, n);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::plusDiag(double a, double b) {
	Dense*I = new Dense;
	I->initial(m, 1);
	I->setValue(1.0f);
	plusDiag(I, a, b);
	delete I;
}

void Dense::solve(Dense*x) {
	int size = 0;
	double* buffer = 0;
	double* tau = 0;
	int*info = 0;
	int h_info = 0;
	double one = 1.0f;
	checkCudaErrors(hipMalloc((void** ) &info, sizeof(int)));
	checkCudaErrors(hipMalloc((void** ) &tau, sizeof(double) * m));
	checkCudaErrors(
			hipsolverDnDgeqrf_bufferSize(solver_handle, m, m, cu_val, m, &size));
	checkCudaErrors(hipMalloc((void** ) &buffer, sizeof(double) * size));
	checkCudaErrors(
			hipsolverDnDgeqrf(solver_handle, m, m, cu_val, m, tau, buffer, size,
					info));
	checkCudaErrors(
			hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));
	if (0 != h_info) {
		fprintf(stderr, "Error: LU factorization failed\n");
	}
//	x->initial(b->m, b->n);
//	checkCudaErrors(
//			hipMemcpy(x->cu_val, b->cu_val, sizeof(double) * b->length(),
//					hipMemcpyDeviceToDevice));
	checkCudaErrors(
			hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, x->m,
					x->n, m, cu_val, m, tau, x->cu_val, x->m, buffer, size,
					info));
	checkCudaErrors(
			hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, x->m, x->n, &one, cu_val, m, x->cu_val, x->m));
	checkCudaErrors(hipFree(buffer));
	checkCudaErrors(hipFree(tau));
	checkCudaErrors(hipFree(info));
}

void Dense::scale(double a) {
	int len = length();
	checkCudaErrors(hipblasDscal(handle, len, &a, cu_val, 1));
}

double Dense::trace(Dense*d) {
	Dense* r = new Dense;
	transpose(r);
	r->eTimes(d);
	Dense* v = new Dense;
	r->rowSum(v);
	double sum = v->sum();
	delete r;
	delete v;
	return sum;
}

double Dense::sum() {
	Dense* vec = new Dense;
	rowSum(vec);
	Dense* ones = new Dense;
	ones->initial(m, 1);
	ones->setValue(1.0f);
	double s = vec->dot(ones);
	delete vec;
	delete ones;
	return s;
}

double Dense::frobenius() {
	double n = norm2();
	return n * n;
}

double Dense::norm2() {
	double n2;
//hipblasSdot(handle, m, cu_val, 1, cu_val, 1, &n2);
	checkCudaErrors(hipblasDnrm2(handle, length(), cu_val, 1, &n2));
	return n2;
}

double Dense::norm1() {
	double n1;
	checkCudaErrors(hipblasDasum(handle, length(), cu_val, 1, &n1));
	return n1;
}

double Dense::square() {
	double n = norm2();
	return n * n;
}

void Dense::getCol(Dense*d, int c) {
	d->clean();
	d->initial(m, 1);
	checkCudaErrors(
			hipMemcpy(d->cu_val, &cu_val[c * m], sizeof(double) * m,
					hipMemcpyDeviceToDevice));
}

void Dense::setCol(Dense*d, int c) {
	checkCudaErrors(
			hipMemcpy(&cu_val[c * m], d->cu_val, sizeof(double) * m,
					hipMemcpyDeviceToDevice));
}

thrust::device_ptr<int> Dense::sortKeyCol(bool greater) {
	int len = m * n;
	thrust::device_ptr<int> order = thrust::device_malloc<int>(len);
	thrust::device_vector<int> d(len);
	thrust::device_vector<int> seg(len);
	thrust::sequence(order, order + len);
	thrust::fill(d.begin(), d.end(), m);
	if (greater)
		thrust::stable_sort_by_key(val, val + len, order,
				thrust::greater<double>());
	else
		thrust::stable_sort_by_key(val, val + len, order);
	thrust::transform(order, order + len, d.begin(), seg.begin(),
			thrust::divides<int>());
	thrust::transform(order, order + len, d.begin(), order,
			thrust::modulus<int>());
	thrust::stable_sort_by_key(seg.begin(), seg.end(), order);
	return order;
}

thrust::device_ptr<int> Dense::sortKeyRow(bool greater) {
	Dense* T = new Dense;
	this->transpose(T);
	thrust::device_ptr<int> order = T->sortKeyCol(greater);
	delete T;
	return order;
}

void Dense::getRow(Dense*d, int r) {
	d->clean();
	d->initial(1, n);
	int thread = THREADS;
	int block = (n + thread - 1) / thread;
	getRowKernel<<<block, thread>>>(cu_val, d->cu_val, r, m, n);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::setRow(Dense*d, int r) {
	int thread = THREADS;
	int block = (n + thread - 1) / thread;
	setRowKernel<<<block, thread>>>(cu_val, d->cu_val, r, m, n);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::diag(Dense*d) {
	d->clean();
	if (m == 1 || n == 1) {
		int l = m * n;
		d->initial(l, l);
		d->setValue(0.0f);
		for (int i = 0; i < l; ++i)
			d->val[i * l + i] = val[i];
	} else {
		d->initial(m, 1);
		for (int i = 0; i < m; ++i)
			d->val[i] = val[i * m + i];
	}
}

void Dense::rbind(Dense*r, Dense*d, double a, double b) {
	if (n != d->n) {
		printf("column not match\n");
		return;
	}
	r->clean();
	r->initial(m + d->m, n);
	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
	dim3 numBlocks((r->m + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(r->n + threadsPerBlock.y - 1) / threadsPerBlock.y);
	rbindKernel<<<numBlocks, threadsPerBlock>>>(r->cu_val, cu_val, d->cu_val, m,
			d->m, n, a, b);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::rbind(Dense*d, double a, double b) {
	Dense*r = new Dense;
	rbind(r, d, a, b);
	r->clone(this);
	delete r;
}

void Dense::cbind(Dense*r, Dense*d, double a, double b) {
	if (m != d->m) {
		printf("row not match\n");
		return;
	}
	r->clean();
	r->initial(m, n + d->n);
//	checkCudaErrors(hipblasDcopy(handle, m*n, cu_val, 1, r->cu_val, 1));
//	checkCudaErrors(
//			hipblasDcopy(handle, m*d->n, d->cu_val, 1, r->cu_val + m*n, 1));
	thrust::transform(val, val + length(), r->val, multiply_const(a));
	thrust::transform(d->val, d->val + d->length(), r->val + m * n,
			multiply_const(b));
//	thrust::transform(r->val, r->val + m * n, multiply_const(a));
//	thrust::transform(r->val + m * n, r->val + m * r->n, multiply_const(b));
}

void Dense::cbind(Dense*d, double a, double b) {
	Dense*r = new Dense;
	cbind(r, d, a, b);
	r->clone(this);
	delete r;
}

Dense::~Dense() {
//printf("dense destructor called\n");
	clean();
}

void Sparse::readCSR(const char* filename) {
	FILE* file = fopen(filename, "r");
	char line[100];
	fgets(line, 100, file);
	fscanf(file, "%d %d %d", &m, &n, &nnz);
	initialBoth();
	int i, r, c;
	double v;
	size_t size = sizeof(int) * nnz;
	int*row_host = (int*) malloc(size);
	int*col_host = (int*) malloc(size);
	double*val_host = (double*) malloc(sizeof(double) * nnz);
	if (strstr(line, "pattern") != NULL) {
		for (i = 0; i < nnz; ++i) {
			fscanf(file, "%d %d", &r, &c);
			row_host[i] = r - 1;
			col_host[i] = c - 1;
			val_host[i] = 1;
		}
	} else {
		for (i = 0; i < nnz; ++i) {
			fscanf(file, "%d %d %lf", &r, &c, &v);
			row_host[i] = r - 1;
			col_host[i] = c - 1;
			val_host[i] = v;
		}
	}
	fclose(file);
	uploadCSR(row_host, col_host, val_host);
	delete row_host;
	delete col_host;
	delete val_host;
}

void Sparse::setDiag(Dense*d) {
	nnz = m = n = d->length();
	initialBoth();
	size_t size = sizeof(int) * nnz;
	int*row_host = (int*) malloc(size);
	int*col_host = (int*) malloc(size);
	double*val_host = (double*) malloc(sizeof(double) * nnz);
	for (int i = 0; i < nnz; ++i) {
		row_host[i] = col_host[i] = i;
		val_host[i] = d->val[i];
	}
	uploadCSR(row_host, col_host, val_host);
	delete row_host;
	delete col_host;
	delete val_host;
}

void Sparse::setIdentity(int s) {
	nnz = m = n = s;
	initialBoth();
	size_t size = sizeof(int) * nnz;
	int*row_host = (int*) malloc(size);
	int*col_host = (int*) malloc(size);
	double*val_host = (double*) malloc(sizeof(double) * nnz);
	for (int i = 0; i < nnz; ++i) {
		row_host[i] = col_host[i] = i;
		val_host[i] = 1;
	}
	uploadCSR(row_host, col_host, val_host);
	delete row_host;
	delete col_host;
	delete val_host;
}

void Sparse::readCSC(const char* filename) {
	FILE* file = fopen(filename, "r");
	char line[100];
	fgets(line, 100, file);
	fscanf(file, "%d %d %d", &m, &n, &nnz);
	initialBoth();
	int i, r, c;
	double v;
	size_t size = sizeof(int) * nnz;
	int*row_host = (int*) malloc(size);
	int*col_host = (int*) malloc(size);
	double*val_host = (double*) malloc(sizeof(double) * nnz);
	if (strstr(line, "pattern") != NULL) {
		for (i = 0; i < nnz; ++i) {
			fscanf(file, "%d %d", &r, &c);
			row_host[i] = r - 1;
			col_host[i] = c - 1;
			val_host[i] = 1;
		}
	} else {
		for (i = 0; i < nnz; ++i) {
			fscanf(file, "%d %d %lf", &r, &c, &v);
			row_host[i] = r - 1;
			col[i] = c - 1;
			val[i] = v;
		}
	}
	fclose(file);
	uploadCSC(row_host, col_host, val_host);
	delete row_host;
	delete col_host;
	delete val_host;
}

void Sparse::writeCSR(const char* filename) {
	std::ofstream out(filename);
	out << "%%MatrixMarket matrix coordinate real general" << std::endl;
	out << m << " " << n << " " << nnz << std::endl;
	for (int i = 0; i < m; ++i) {
		int nnz = row[i + 1] - row[i];
		for (int j = 0; j < nnz; ++j)
			out << i + 1 << " " << col[row[i] + j] + 1 << " " << val[row[i] + j]
					<< std::endl;
	}
	out.close();
}

void Sparse::writeCSC(const char* filename) {
	std::ofstream out(filename);
	thrust::device_ptr<int> row(cu_row);
	thrust::device_ptr<int> col(cu_col_index);
	thrust::device_ptr<double> val(trans_val);
	out << "%%MatrixMarket matrix coordinate real general" << std::endl;
	out << m << " " << n << " " << nnz << std::endl;
	for (int i = 0; i < n; ++i) {
		int nnz = col[i + 1] - col[i];
		for (int j = 0; j < nnz; ++j)
			out << row[col[i] + j] + 1 << " " << i + 1 << " " << val[col[i] + j]
					<< std::endl;
	}
	out.close();
}

void Sparse::rowSum(Dense*d) {
	d->clean();
	d->initial(m, 1);
	Dense*one = new Dense;
	one->initial(n, 1);
	one->setValue(1.0f);
	checkCudaErrors(
			hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz,
					&ONE, descr, cu_val, cu_row_index, cu_col, one->cu_val,
					&ZERO, d->cu_val));
	delete one;
}

void Sparse::csrmm2(Dense*r, Dense*d, bool transA, bool transB, double a,
		double b) {
	int m = r->m;
	int n = r->n;
	int k = transA ? this->m : this->n;
	if (!transB) {
		Dense* trans = new Dense;
		d->transpose(trans);
		if (transA) {
			checkCudaErrors(
					hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
							HIPSPARSE_OPERATION_TRANSPOSE, m, n, k, nnz, &a,
							descr, trans_val, cu_col_index, cu_row,
							trans->cu_val, n, &b, r->cu_val, m));
		} else {
			checkCudaErrors(
					hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
							HIPSPARSE_OPERATION_TRANSPOSE, m, n, k, nnz, &a,
							descr, cu_val, cu_row_index, cu_col, trans->cu_val,
							n, &b, r->cu_val, m));
		}
		delete trans;
	} else {
		if (transA) {
			checkCudaErrors(
					hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
							HIPSPARSE_OPERATION_TRANSPOSE, m, n, k, nnz, &a,
							descr, trans_val, cu_col_index, cu_row, d->cu_val,
							n, &b, r->cu_val, m));
		} else {
			checkCudaErrors(
					hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
							HIPSPARSE_OPERATION_TRANSPOSE, m, n, k, nnz, &a,
							descr, cu_val, cu_row_index, cu_col, d->cu_val, n,
							&b, r->cu_val, m));
		}
	}
}

void Sparse::csrmv(Dense*y, Dense*x, double alpha, double beta, bool trans) {
	if (trans) {
		checkCudaErrors(
				hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, m, n, nnz,
						&alpha, descr, cu_val, cu_row_index, cu_col, x->cu_val,
						&beta, y->cu_val));
	} else {
		checkCudaErrors(
				hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n,
						nnz, &alpha, descr, cu_val, cu_row_index, cu_col,
						x->cu_val, &beta, y->cu_val));
	}
}

void Sparse::inv(Dense* d) {
	csrilu02Info_t info_LU;
	int size;
	int*buffer;
	int position;
	checkCudaErrors(hipsparseCreateCsrilu02Info(&info_LU));
	checkCudaErrors(
			hipsparseDcsrilu02_bufferSize(handle, m, nnz, descr, cu_val,
					cu_row_index, cu_col, info_LU, &size));
	checkCudaErrors(hipMalloc((void** ) &buffer, size));
	checkCudaErrors(
			hipsparseDcsrilu02_analysis(handle, m, nnz, descr, cu_val,
					cu_row_index, cu_col, info_LU,
					HIPSPARSE_SOLVE_POLICY_NO_LEVEL, &buffer));
	checkCudaErrors(hipsparseXcsrilu02_zeroPivot(handle, info_LU, &position));
	checkCudaErrors(
			hipsparseDcsrilu02(handle, m, nnz, descr, cu_val, cu_row_index,
					cu_col, info_LU, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer));
	checkCudaErrors(hipsparseDestroyCsrilu02Info(info_LU));
	checkCudaErrors(hipFree(buffer));

	Dense* t = new Dense;
	t->initial(m, m);
	d->clean();
	d->initial(m, m);
	cusparseSolveAnalysisInfo_t info;
	checkCudaErrors(cusparseCreateSolveAnalysisInfo(&info));
	checkCudaErrors(
			cusparseDcsrsm_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m,
					nnz, descr_L, cu_val, cu_row_index, cu_col, info));
	Dense* I = new Dense;
	I->setIdentity(m);
	checkCudaErrors(
			cusparseDcsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, m,
					&ONE, descr_L, cu_val, cu_row_index, cu_col, info,
					I->cu_val, m, t->cu_val, m));
	checkCudaErrors(
			cusparseDcsrsm_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m,
					nnz, descr_U, cu_val, cu_row_index, cu_col, info));
	checkCudaErrors(
			cusparseDcsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, m,
					&ONE, descr_U, cu_val, cu_row_index, cu_col, info,
					t->cu_val, m, d->cu_val, m));
	checkCudaErrors(cusparseDestroySolveAnalysisInfo(info));
	delete I;
	delete t;
}

void Sparse::pinv(Dense*r, double tol) {
	Dense* d = new Dense;
	toDense(d);
	d->pinv(r, tol);
	delete d;
}

void Sparse::times(Dense* r, Dense* d, bool transA, bool transB) {
	r->clean();
	int m = transA ? this->n : this->m;
	int n = transB ? d->m : d->n;
	r->initial(m, n);
	csrmm2(r, d, transA, transB, 1.0f, 0.0f);
}

void Sparse::plus(Dense*r, double a, double b, bool trans) {
	Dense*I = new Dense;
	if (trans)
		I->setIdentity(m);
	else
		I->setIdentity(n);
	csrmm2(r, I, trans, false, a, b);
	delete I;
}

void Sparse::plus(Sparse*r, Sparse* s, double a, double b) {
	r->m = m;
	r->n = n;
	r->nnz = 0;
	r->clean();
	checkCudaErrors(
			hipMalloc((void** ) &r->cu_row_index, sizeof(int) * (r->m + 1)));
	int *nnzTotalDevHostPtr = &s->nnz;
	checkCudaErrors(
			hipsparseXcsrgeamNnz(handle, m, n, descr, nnz, cu_row_index, cu_col,
					s->descr, s->nnz, s->cu_row_index, s->cu_col, r->descr,
					r->cu_row_index, nnzTotalDevHostPtr));
	if (NULL != nnzTotalDevHostPtr)
		r->nnz = *nnzTotalDevHostPtr;
	else {
		int base;
		checkCudaErrors(
				hipMemcpy(&r->nnz, r->cu_row_index + m, sizeof(int),
						hipMemcpyDeviceToHost));
		checkCudaErrors(
				hipMemcpy(&base, r->cu_row_index, sizeof(int),
						hipMemcpyDeviceToHost));
		r->nnz -= base;
	}
	r->initialBoth();
	checkCudaErrors(
			hipsparseDcsrgeam(handle, m, n, &a, descr, nnz, cu_val, cu_row_index,
					cu_col, &b, s->descr, s->nnz, s->cu_val, s->cu_row_index,
					s->cu_col, r->descr, r->cu_val, r->cu_row_index,
					r->cu_col));
//hipsparseScsrgeam(handle,m,n,&a,descr,nnz,)
}

void Sparse::times(Sparse*nmat, Sparse*mat, bool transA, bool transB) {
	int m, n, k;
	if (!transA) {
		m = this->m;
		k = this->n;
	} else {
		m = this->n;
		k = this->m;
		transpose();
	}
	if (!transB)
		n = mat->n;
	else {
		n = mat->m;
		mat->transpose();
	}
	nmat->m = m;
	nmat->n = n;
	nmat->nnz = 0;
	nmat->clean();
	checkCudaErrors(
			hipMalloc((void** ) &nmat->cu_row_index,
					sizeof(int) * (nmat->m + 1)));
	int *nnzTotalDevHostPtr = &nmat->nnz;
	if (!transA && !transB)
		checkCudaErrors(
				hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						cu_row_index, cu_col, descr, mat->nnz,
						mat->cu_row_index, mat->cu_col, descr,
						nmat->cu_row_index, nnzTotalDevHostPtr));
	else if (transA && !transB)
		checkCudaErrors(
				hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						cu_col_index, cu_row, descr, mat->nnz,
						mat->cu_row_index, mat->cu_col, descr,
						nmat->cu_row_index, nnzTotalDevHostPtr));
	else if (!transA && transB)
		checkCudaErrors(
				hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						cu_row_index, cu_col, descr, mat->nnz,
						mat->cu_col_index, mat->cu_row, descr,
						nmat->cu_row_index, nnzTotalDevHostPtr));
	else
		checkCudaErrors(
				hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						cu_col_index, cu_row, descr, mat->nnz,
						mat->cu_col_index, mat->cu_row, descr,
						nmat->cu_row_index, nnzTotalDevHostPtr));

	if (NULL != nnzTotalDevHostPtr)
		nmat->nnz = *nnzTotalDevHostPtr;
	else {
		int base;
		checkCudaErrors(
				hipMemcpy(&nmat->nnz, nmat->cu_row_index + m, sizeof(int),
						hipMemcpyDeviceToHost));
		checkCudaErrors(
				hipMemcpy(&base, nmat->cu_row_index, sizeof(int),
						hipMemcpyDeviceToHost));
		nmat->nnz -= base;
	}
	nmat->initialBoth();
	if (!transA && !transB)
		checkCudaErrors(
				hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						cu_val, cu_row_index, cu_col, mat->descr, mat->nnz,
						mat->cu_val, mat->cu_row_index, mat->cu_col,
						nmat->descr, nmat->cu_val, nmat->cu_row_index,
						nmat->cu_col));
	else if (transA && !transB)
		checkCudaErrors(
				hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						trans_val, cu_col_index, cu_row, mat->descr, mat->nnz,
						mat->cu_val, mat->cu_row_index, mat->cu_col,
						nmat->descr, nmat->cu_val, nmat->cu_row_index,
						nmat->cu_col));
	else if (!transA && transB)
		checkCudaErrors(
				hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						cu_val, cu_row_index, cu_col, mat->descr, mat->nnz,
						mat->trans_val, mat->cu_col_index, mat->cu_row,
						nmat->descr, nmat->cu_val, nmat->cu_row_index,
						nmat->cu_col));
	else
		checkCudaErrors(
				hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
						HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descr, nnz,
						trans_val, cu_col_index, cu_row, mat->descr, mat->nnz,
						mat->trans_val, mat->cu_col_index, mat->cu_row,
						nmat->descr, nmat->cu_val, nmat->cu_row_index,
						nmat->cu_col));
}

void Sparse::rowNorm() {
	Dense*d = new Dense;
	rowNorm(d);
	delete d;
}

void Sparse::clean() {
	if (cu_row)
		checkCudaErrors(hipFree(cu_row));
	if (cu_col)
		checkCudaErrors(hipFree(cu_col));
	if (cu_val)
		checkCudaErrors(hipFree(cu_val));
	if (trans_val)
		checkCudaErrors(hipFree(trans_val));
	if (cu_row_index)
		checkCudaErrors(hipFree(cu_row_index));
	if (cu_col_index)
		checkCudaErrors(hipFree(cu_col_index));
	cu_row = 0;
	cu_col = 0;
	cu_val = 0;
	trans_val = 0;
	cu_row_index = 0;
	cu_col_index = 0;
}

void Sparse::initialCSR() {
//printf("m=%d,n=%d,nnz=%d\n", m, n,nnz);
	if (nnz > 0) {
		if (!cu_row)
			checkCudaErrors(hipMalloc((void** ) &cu_row, sizeof(int) * nnz));
		if (!cu_col)
			checkCudaErrors(hipMalloc((void** ) &cu_col, sizeof(int) * nnz));
		if (!cu_val)
			checkCudaErrors(
					hipMalloc((void** ) &cu_val, sizeof(double) * nnz));
	}
	if (m > 0 && !cu_row_index)
		checkCudaErrors(
				hipMalloc((void** ) &cu_row_index, sizeof(int) * (m + 1)));
}

void Sparse::initialCSC() {
//printf("m=%d,n=%d,nnz=%d\n", m, n, nnz);
	if (nnz > 0) {
		if (!cu_row)
			checkCudaErrors(hipMalloc((void** ) &cu_row, sizeof(int) * nnz));
		if (!cu_col)
			checkCudaErrors(hipMalloc((void** ) &cu_col, sizeof(int) * nnz));
		if (!trans_val)
			checkCudaErrors(
					hipMalloc((void** ) &trans_val, sizeof(double) * nnz));
	}
	if (n > 0 && !cu_col_index)
		checkCudaErrors(
				hipMalloc((void** ) &cu_col_index, sizeof(int) * (n + 1)));
}

void Sparse::initialBoth() {
//printf("m=%d,n=%d,nnz=%d\n", m, n, nnz);
	if (nnz > 0) {
		if (!cu_row)
			checkCudaErrors(hipMalloc((void** ) &cu_row, sizeof(int) * nnz));
		if (!cu_col)
			checkCudaErrors(hipMalloc((void** ) &cu_col, sizeof(int) * nnz));
		if (!cu_val)
			checkCudaErrors(
					hipMalloc((void** ) &cu_val, sizeof(double) * nnz));
		if (!trans_val)
			checkCudaErrors(
					hipMalloc((void** ) &trans_val, sizeof(double) * nnz));
	}
	if (m > 0 && !cu_row_index)
		checkCudaErrors(
				hipMalloc((void** ) &cu_row_index, sizeof(int) * (m + 1)));
	if (n > 0 && !cu_col_index)
		checkCudaErrors(
				hipMalloc((void** ) &cu_col_index, sizeof(int) * (n + 1)));
	row = thrust::device_pointer_cast(cu_row_index);
	col = thrust::device_pointer_cast(cu_col);
	val = thrust::device_pointer_cast(cu_val);
}

void Sparse::outerTimes(Sparse* s) {
	times(s, this, true, false);
}

void Sparse::outerTimes(Dense*B, Dense*A) {
	Dense*T = new Dense;
	times(T, A, true, false);
	times(B, T, true, true);
	B->transpose();
	delete T;
}

void Sparse::innerTimes(Sparse* s) {
	times(s, this, false, true);
}

void Sparse::innerTimes(Dense*B, Dense*A) {
	Dense*T = new Dense;
	times(T, A, false, false);
	times(B, T, false, true);
	B->transpose();
	delete T;
}

void Sparse::uploadCSR(int* row, int* col, double* val) {
	checkCudaErrors(
			hipMemcpy(cu_row, row, nnz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(cu_col, col, nnz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(cu_val, val, nnz * sizeof(double),
					hipMemcpyHostToDevice));
	checkCudaErrors(
			hipsparseXcoo2csr(handle, cu_row, nnz, m, cu_row_index,
					HIPSPARSE_INDEX_BASE_ZERO));
	checkCudaErrors(
			hipsparseDcsr2csc(handle, m, n, nnz, cu_val, cu_row_index, cu_col,
					trans_val, cu_row, cu_col_index, HIPSPARSE_ACTION_NUMERIC,
					HIPSPARSE_INDEX_BASE_ZERO));
}

void Sparse::uploadCSC(int* row, int* col, double* val) {
	checkCudaErrors(
			hipMemcpy(cu_row, row, nnz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(cu_col, col, nnz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(trans_val, val, nnz * sizeof(double),
					hipMemcpyHostToDevice));
	checkCudaErrors(
			hipsparseXcoo2csr(handle, cu_col, nnz, n, cu_col_index,
					HIPSPARSE_INDEX_BASE_ZERO));
	checkCudaErrors(
			hipsparseDcsr2csc(handle, n, m, nnz, trans_val, cu_col_index, cu_row,
					cu_val, cu_col, cu_row_index, HIPSPARSE_ACTION_NUMERIC,
					HIPSPARSE_INDEX_BASE_ZERO));
}

void Sparse::diagTimes(Sparse*res, Dense* diag, bool trans) {
	Sparse* diagView = new Sparse;
	diagView->setDiag(diag);
	times(res, diagView, trans, false);
	delete diagView;
}

void Sparse::print() {
	std::cout << "%%MatrixMarket matrix coordinate real general" << std::endl;
	std::cout << m << " " << n << " " << nnz << std::endl;
	for (int i = 0; i < m; ++i) {
		int nnz = row[i + 1] - row[i];
		for (int j = 0; j < nnz; ++j)
			std::cout << i + 1 << " " << col[row[i] + j] + 1 << " "
					<< val[row[i] + j] << std::endl;
	}
}

void Sparse::printFull() {
	Dense*d = new Dense;
	toDense(d);
	d->print();
	delete d;
}

void Sparse::transpose() {
	if (!cu_col_index) {
		checkCudaErrors(
				hipMalloc((void** ) &cu_col_index, sizeof(int) * (n + 1)));
		checkCudaErrors(hipMalloc((void** ) &trans_val, sizeof(double) * nnz));
		checkCudaErrors(
				hipsparseDcsr2csc(handle, m, n, nnz, cu_val, cu_row_index,
						cu_col, trans_val, cu_row, cu_col_index,
						HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
	}
}

void Sparse::selfTimes(Dense*r, Dense* d) {
	times(r, d, false, true);
	eTimes(r);
	rowSum(r);
}

void Sparse::toDense(Dense* dense) {
	dense->clean();
	dense->initial(m, n);
	checkCudaErrors(
			hipsparseDcsr2dense(handle, m, n, descr, cu_val, cu_row_index,
					cu_col, dense->cu_val, dense->m));
}

double Sparse::getElem(int i, int j) {
	int nnz = row[i + 1] - row[i];
	int k;
	for (k = 0; k < nnz && col[row[i] + k] < j; ++k)
		;
	if (k < nnz && col[row[i] + k] == j)
		return val[row[i] + k];
	return 0.0f;
}

Sparse::~Sparse() {
//printf("sparse destructor called\n");
	clean();
}

void Sparse::rowVec(Dense*d, int r, bool column) {
	d->clean();
	if (column)
		d->initial(n, 1);
	else
		d->initial(1, n);
	d->setValue(0.0f);
	int thread = THREADS;
	int block = (n + thread - 1) / thread;
	subvecKernel<<<block, thread>>>(cu_row_index, cu_col, r, cu_val, d->cu_val);
	checkCudaErrors(hipDeviceSynchronize());
}

void Sparse::rowVec(Dense*d, int r) {
	rowVec(d, r, false);
}

void Sparse::colVec(Dense*d, int c, bool row) {
	d->clean();
	if (row)
		d->initial(1, m);
	else
		d->initial(m, 1);
	d->setValue(0.0f);
	int thread = THREADS;
	int block = (m + thread - 1) / thread;
	subvecKernel<<<block, thread>>>(cu_col_index, cu_row, c, trans_val,
			d->cu_val);
	checkCudaErrors(hipDeviceSynchronize());
}

void Sparse::colVec(Dense*d, int c) {
	colVec(d, c, false);
}

void Sparse::rowNorm(Dense*r) {
	r->clean();
	r->initial(m, 1);
	int thread = THREADS;
	int block = (m + thread - 1) / thread;
	rowNormSPKernel<<<block, thread>>>(cu_row_index, cu_val, r->cu_val, m);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(
			hipsparseDcsr2csc(handle, m, n, nnz, cu_val, cu_row_index, cu_col,
					trans_val, cu_row, cu_col_index, HIPSPARSE_ACTION_NUMERIC,
					HIPSPARSE_INDEX_BASE_ZERO));
}

void Sparse::rowMultiply(double alpha) {
	int thread = THREADS;
	int block = (m + thread - 1) / thread;
	rowMulKernel<<<block, thread>>>(cu_row_index, cu_val, alpha, m);
	checkCudaErrors(hipDeviceSynchronize());
}

void Sparse::colSum(Dense*d) {
	transpose();
	d->clean();
	d->initial(1, n);
	int thread = THREADS;
	int block = (m + thread - 1) / thread;
	rowSumKernel<<<block, thread>>>(cu_col_index, trans_val, n, d->cu_val);
	checkCudaErrors(hipDeviceSynchronize());
}

void Sparse::eTimes(Dense* d) {
	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
	dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(n + threadsPerBlock.y - 1) / threadsPerBlock.y);
	eTimesSPKernel<<<numBlocks, threadsPerBlock>>>(cu_row_index, cu_col, cu_val,
			d->cu_val, m, n);
	checkCudaErrors(hipDeviceSynchronize());
}

void Sparse::eTimes(Dense*r, Sparse*s) {
	int block = (m + THREADS - 1) / THREADS;
	r->clean();
	r->initial(s->n, s->m);
	etimesKernel<<<block, THREADS>>>(cu_row_index, cu_col, cu_val,
			s->cu_row_index, s->cu_col, s->cu_val, r->cu_val, m, n);
	checkCudaErrors(hipDeviceSynchronize());
	r->transpose();
}

void Dense::signPlus(Dense*s, double v) {
	int thread = THREADS;
	int len = length();
	int block = (len + thread - 1) / thread;
	signPlusKernel<<<block, thread>>>(cu_val, s->cu_val, v, len);
	checkCudaErrors(hipDeviceSynchronize());
}
//struct square_root: public thrust::unary_function<float, float> {
//	__host__ __device__
//	float operator()(float x) const {
//		return sqrtf(x);
//	}
//};
void Dense::square_root() {
	pow(0.5f);
}
struct power: public thrust::unary_function<double, double> {
	const double a;
	power(double _a) :
			a(_a) {
	}
	__host__ __device__
	double operator()(const double &x) const {
		return powf(x, a);
	}
};
void Dense::pow(double ind) {
	thrust::transform(val, val + length(), val, power(ind));
}

void Dense::project() {
	int thread = THREADS;
	int len = length();
	int block = (len + thread - 1) / thread;
	projectKernel<<<block, thread>>>(cu_val, len);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::shrink(double tau) {
	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
	dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(n + threadsPerBlock.y - 1) / threadsPerBlock.y);
	shrinkKernel<<<numBlocks, threadsPerBlock>>>(tau, cu_val, m, n);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::repmat(Dense*d, int m, int n) {
	d->clean();
	d->initial(m * this->m, n * this->n);
	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
	dim3 numBlocks((d->m + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(d->n + threadsPerBlock.y - 1) / threadsPerBlock.y);
	repmatKernel<<<numBlocks, threadsPerBlock>>>(d->cu_val, cu_val, d->m, d->n,
			this->m, this->n);
	checkCudaErrors(hipDeviceSynchronize());
}

void Dense::sub(Dense*d, int rs, int re, int cs, int ce) {
	d->clean();
	int m = re - rs;
	int n = ce - cs;
	d->initial(m, n);
	dim3 threadsPerBlock(PER_THREADS, PER_THREADS);
	dim3 numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(n + threadsPerBlock.y - 1) / threadsPerBlock.y);
	subKernel<<<numBlocks, threadsPerBlock>>>(cu_val, d->cu_val, rs, re, cs, ce,
			this->m);
	checkCudaErrors(hipDeviceSynchronize());
}

